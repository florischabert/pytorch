#include "THCTensorTypeUtils.cuh"
#include "THCTensor.h"
#include "THCTensorCopy.h"
#include "THCHalf.h"
#include <stdlib.h>

#define IMPL_TENSOR_UTILS(TENSOR_TYPE, DATA_TYPE)                       \
                                                                        \
TENSOR_TYPE*                                                            \
TensorUtils<TENSOR_TYPE>::newTensor(THCState* state) {                  \
  return TENSOR_TYPE##_new(state);                                      \
}                                                                       \
                                                                        \
TENSOR_TYPE*                                                            \
TensorUtils<TENSOR_TYPE>::newContiguous(THCState* state,                \
                                        TENSOR_TYPE* t) {               \
  return TENSOR_TYPE##_newContiguous(state, t);                         \
}                                                                       \
                                                                        \
void                                                                    \
TensorUtils<TENSOR_TYPE>::freeCopyTo(THCState* state,                   \
                                     TENSOR_TYPE* src,                  \
                                     TENSOR_TYPE* dst) {                \
  TENSOR_TYPE##_freeCopyTo(state, src, dst);                            \
}                                                                       \
                                                                        \
void                                                                    \
TensorUtils<TENSOR_TYPE>::copyIgnoringOverlaps(THCState* state,         \
                                               TENSOR_TYPE* dst,        \
                                               TENSOR_TYPE* src) {      \
  return TENSOR_TYPE##_copyIgnoringOverlaps(state, dst, src);           \
}                                                                       \

IMPL_TENSOR_UTILS(THCudaByteTensor, uint8_t)
IMPL_TENSOR_UTILS(THCudaCharTensor, int8_t)
IMPL_TENSOR_UTILS(THCudaShortTensor, int16_t)
IMPL_TENSOR_UTILS(THCudaIntTensor, int32_t)
IMPL_TENSOR_UTILS(THCudaLongTensor, int64_t)
IMPL_TENSOR_UTILS(THCudaTensor, float)
IMPL_TENSOR_UTILS(THCudaDoubleTensor, double)

#ifdef CUDA_HALF_TENSOR
IMPL_TENSOR_UTILS(THCudaHalfTensor, half);
#endif

#undef IMPL_TENSOR_UTILS
